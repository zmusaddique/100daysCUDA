#include "hip/hip_runtime.h"
__global__ void sumReductionKernel(float *a, float *b){
  int i = 2*threadIdx.x;
  for (int stride =1; stride<Bi.x; stride *= 2){
    if (threadIdx.x % stride == 0){
      a[i] += a[i+stride];
    }
    __syncthreads();
  }
  if (threadIdx.x == 0){
    *b = a[0]; 
  }
}

int main(){
  float *a = malloc(16 * sizeof(float));
  float *b = malloc(sizeof(float));

  for (int i = 0; i<16; i++){
    a[i] = i+1.0; 
  }
   
  float *a_d, *b_d;
  hipMalloc((void **) &a_d, 16 * sizeof(float));
  hipMalloc((void **) &b_d, sizeof(float));

  hipMemcpy(a_d, a, 16*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(b_d, b, sizeof(float), hipMemcpyHostToDevice);
  sumReductionKernel<<<4*4, 4>>>(a, b);

  hipMemcpy(b, b_d, sizeof(float), hipMemcpyDeviceToHost);
  for (int i = 0; i<16; i++){
    printf("%.2f", a[i]);  
       
  }
       printf("%.2f", b);
}
