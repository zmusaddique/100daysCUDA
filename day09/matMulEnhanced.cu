
#include <hip/hip_runtime.h>
#include <stdio.h>

#define TILE_SIZE 2

__global__ void matmul_generalized(float *A, float *B, float *R, int width) {
 
  extern __shared__ float A_B_s[];

  int tx = threadIdx.x; int ty = threadIdx.y;
  int bx = blockIdx.x; int by = blockIdx.y;

  int Row = by * blockDim.y + ty;
  int Col = bx * blockDim.x + tx;

  float Rval = 0.0f;

  for (int ph = 0; ph < (width + TILE_SIZE - 1)/TILE_SIZE; ph++){

    int ACol = ph * TILE_SIZE + tx;
    if (Row < width && ACol < width)
      A_B_s[ty*TILE_SIZE + tx] = A[Row * width + ACol];
    else 
      A_B_s[ty*TILE_SIZE + tx] = 0.0f;

    int BRow = ph  * TILE_SIZE + ty;
    if (Col < width && BRow < width)
      A_B_s[TILE_SIZE*TILE_SIZE + ty * TILE_SIZE + tx]= B[BRow * width + Col];
    else
      A_B_s[TILE_SIZE*TILE_SIZE + ty * TILE_SIZE + tx] = 0.0f; 
    __syncthreads(); // read-after-write dependency 

    for (int i = 0; i<TILE_SIZE; i++){
      Rval+=A_B_s[ty*TILE_SIZE + i] * A_B_s[TILE_SIZE*TILE_SIZE + i * TILE_SIZE + tx];    
    }
    __syncthreads(); // write-after-read dependency
  }
 
  if (Row<width && Col < width)     
    R[Row*width + Col] = Rval;  
}

int main(){
 int N = 3;
  float *A, *B, *R;
  
  int size = N*N * sizeof(float);

  A = (float *)malloc(size);
  B = (float *)malloc(size);
  R = (float *)malloc(size);

  for (int i = 0; i< N; i++){
    for (int j = 0; j<N; j++){
      A[i*N + j] = 1.0f;
      B[i*N + j] = 2.0f;
      R[i*N + j] = 0.0f;
    }
  }

  float *A_d, *B_d, *R_d;
 
  hipMalloc((void **)&A_d, size);
  hipMalloc((void **)&B_d, size);
  hipMalloc((void **)&R_d, size);

  hipMemcpy(A_d, A, size, hipMemcpyHostToDevice); 
  hipMemcpy(B_d, B, size, hipMemcpyHostToDevice); 
  hipMemcpy(R_d, R, size, hipMemcpyHostToDevice); 

  dim3 blockDim(TILE_SIZE, TILE_SIZE);
  dim3 gridDim((N+blockDim.x-1)/blockDim.x, (N+blockDim.y-1)/blockDim.y);

  size_t sharedMemSize = 2 * TILE_SIZE * TILE_SIZE * sizeof(float); 
  matmul_generalized<<<gridDim, blockDim, sharedMemSize>>>(A_d, B_d, R_d, N); 

  hipMemcpy(R, R_d, size, hipMemcpyDeviceToHost);


  hipFree(A_d);
  hipFree(B_d);
  hipFree(R_d);

  // Result
  printf("The resultant matrix R is: \n");
  for (int i = 0; i< N; i++){
    for (int j=0; j < N; j++){
      printf("%.2f ", R[i*N +j]);
    }
    printf("\n");
  }

  printf("The first matrix A was: \n");
  for (int i = 0; i< N; i++){
    for (int j=0; j < N; j++){
      printf("%.2f ", A[i*N +j]);
    }
    printf("\n");
  }

  printf("The second matrix B was: \n");
  for (int i = 0; i< N; i++){
    for (int j=0; j < N; j++){
      printf("%.2f ", B[i*N +j]);
    }
    printf("\n");
  }

  free(A);
  free(B);
  free(R);
}
