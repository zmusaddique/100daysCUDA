
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__
void vecAddKernel(float *A_d, float* B_d, float* C_d, int n){
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  
  if (i<n){
    C_d[i]  = A_d[i] + B_d[i];
  }
}  

void vedAdd(float *A_h, float * B_h, float *C_h, int n){
  // Part 1: Allocate the mem in the device
  //         Copy from host mem to device mem
  int size = n * sizeof(float);
  float *A_d, *B_d, *C_d;

  hipMalloc((void**)&A_d, size);
  hipMalloc((void**)&B_d, size);
  hipMalloc((void**)&C_d, size);

  hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
  hipMemcpy(B_d, B_h, size, hipMemcpyHostToDevice);

  // Part 2: Call kernel & compute 
  vecAddKernel<<<ceil(n/256.0), 256>>>(A_d, B_d, C_d, n);

  hipMemcpy(C_h, C_d, size, hipMemcpyDeviceToHost);

  //Part3: Free the memory in device
  hipFree(A_d);
  hipFree(B_d);
  hipFree(C_d);  
}

int main(){
  float A[] = {1.0, 2.0, 3.0};
  float B[] = {1.0, 2.0 , 3.0};
  float C[3];
  int n = 3;

  vedAdd(A, B, C, n);

  for(int i = 0; i<n; i++){
    printf("%f ", C[i])  ;
  }

  return 0;
}

